// Very minimal skeleton for the kernel


#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void convolution_layer(double input_data[100][100], double filters[10][5][5], double conv_output[10][20][20]) {
    int layer_idx = blockIdx.x;
    int section_x = threadIdx.x;
    int section_y = threadIdx.y;

    // Compute dot product for the 5x5 region
    double dp = 0;
    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            dp += filters[layer_idx][i][j] * input_data[section_x*5 + i][section_y*5 + j];
            //printf("%.2f * %.2f = %.2f\n", filters[layer_idx][i][j], input_data[section_x*5 + i][section_y*5 + j], dp);
        }
    }

    // Update output matrix
    conv_output[layer_idx][section_x][section_y] = dp;
    
    // printf("Layer: %d, section_x: %d, section_y: %d, %.2f\n", layer_idx, section_x, section_y, dp);
}

__device__ void relu_layer(double conv_output[10][20][20]) {
    int layer_idx = blockIdx.x;
    int section_x = threadIdx.x;
    int section_y = threadIdx.y;
    if (conv_output[layer_idx][section_x][section_y] < 0.0) {
        conv_output[layer_idx][section_x][section_y] = 0.0;
    }
    
    // printf("Layer: %d, section_x: %d, section_y: %d, %.2f\n", layer_idx, section_x, section_y, conv_output[layer_idx][section_x][section_y]);
}

__device__ void output_layer_multiply(double conv_output[10][20][20], double weights[10][4000]) {
    int layer_idx = blockIdx.x;
    int section_x = threadIdx.x;
    int section_y = threadIdx.y;
    for (int output_layer_idx = 0; output_layer_idx < 10; output_layer_idx++) {
        weights[output_layer_idx][section_y + section_x*20 + layer_idx*400] = conv_output[layer_idx][section_x][section_y] * weights[output_layer_idx][section_y + section_x*20 + layer_idx*400];
    }

    // printf("output idx: 9, weight idx: %d, %.2f\n", section_y + section_x*20 + layer_idx*400, weights[9][section_y + section_x*20 + layer_idx*400]);
}

__device__ void output_layer_add_1(double weights_multiplication_output[10][4000], double output_layer_temp_1[10][200]) {
    int layer_idx = blockIdx.x;     // 0 - 9
    int section_x = threadIdx.x;    // 0 - 19
    int section_y = threadIdx.y;    // 0 - 19

    // A single block will be responsible for a single output, and then we will split up the summation between 200 threads (per block), each thread summing 20 elements
    if (section_y < 10) {
        // printf("Layer: %d, section_x: %d, section_y: %d\n", layer_idx, section_x, section_y);
        
        double sum = 0;
        for (int i = 0; i < 20; i++) {
            sum += weights_multiplication_output[layer_idx][section_x*200 + section_y*20 + i];
            
            // if (layer_idx == 9 && section_x == 12 && section_y == 9) {
            //     printf("output idx: %d, i: %d, %.2f\n", layer_idx, section_x*200 + section_y*20 + i, weights_multiplication_output[layer_idx][section_x*200 + section_y*20 + i]);
            // }
        }
        output_layer_temp_1[layer_idx][section_x*10 + section_y] = sum;
        
        // if (layer_idx == 9 && section_x == 12 && section_y == 9) {
        //     printf("sum: %.2f\n", sum);
        //     //printf("sum: %.2f\n", output_layer_temp_1[layer_idx][section_x*10 + section_y]);
        // }

        // printf("%.2f\n", output_layer_temp_1[9][129]);

        //printf("output idx: %d, section_x: %d, section_y: %d, i: %d, %.2f\n", layer_idx, section_x, section_y, section_x*10 + section_y, output_layer_temp_1[layer_idx][section_x*10 + section_y]);
    }
}

__device__ void output_layer_add_2(double output_layer_temp_1[10][200], double output_layer_temp_2[10][10]) {
    int layer_idx = blockIdx.x;     // 0 - 9
    int section_x = threadIdx.x;    // 0 - 19
    int section_y = threadIdx.y;    // 0 - 19

    // A single block will be responsible for a single output, and then we will split up the remaining summation between 10 threads (per block), each thread summing 20 elements
    if (section_x < 10 && section_y == 0) {
        // printf("Layer: %d, section_x: %d, section_y: %d\n", layer_idx, section_x, section_y);

        double sum = 0;
        for (int i = 0; i < 20; i++) {
            sum += output_layer_temp_1[layer_idx][section_x*20 + i];
        }
        output_layer_temp_2[layer_idx][section_x] = sum;
    }
}

__device__ void output_layer_add_3(double output_layer_temp_2[10][10], double output_data[10]) {
    int layer_idx = blockIdx.x;     // 0 - 9
    int section_x = threadIdx.x;    // 0 - 19
    int section_y = threadIdx.y;    // 0 - 19

    // A single block will be responsible for a single output, and then we will have 1 thread (per block) compute the remaining summation.
    if (section_x == 0 && section_y == 0) {
        double sum = 0;
        for (int i = 0; i < 10; i++) {
            sum += output_layer_temp_2[layer_idx][i];
            // printf("output idx: %d, i: %d, %.2f\n", layer_idx, i, output_layer_temp_2[layer_idx][i]);
        }
        output_data[layer_idx] = sum;
        // printf("output idx: %d, sum = %.2f\n", layer_idx, sum);
    }
}

// __device__ void output_layer_add(double output_layer_temp_1[10][200], double output_data[10]) {
//     int layer_idx = blockIdx.x;     // 0 - 9
//     int section_x = threadIdx.x;    // 0 - 19
//     int section_y = threadIdx.y;    // 0 - 19

//     if (section_x == 0 && section_y == 0) {
//         double sum = 0;
//         for (int i = 0; i < 200; i++) {
//             sum += output_layer_temp_1[layer_idx][i];
//         }
//         output_data[layer_idx] = sum;
//     }
// }

extern "C" __global__ void compute(
        double input_data[100][100],
        double filters[10][5][5],
        double weights[10][4000],
        double output_data[10]
        ) {

    // int layer_idx = blockIdx.x;
    // int section_x = threadIdx.x;
    // int section_y = threadIdx.y;
    // printf("%d %d %d %.2f\n", layer_idx, section_x, section_y, input_data[0][0]);

    double conv_output[10][20][20];
    convolution_layer(input_data, filters, conv_output);
    relu_layer(conv_output);
    // Temporary matrices for parallelize summing portion of final dot product (divide and conquer strategy)
    double output_layer_temp_1[10][200];
    double output_layer_temp_2[10][10];
    output_layer_multiply(conv_output, weights); // The output of the multiplication step is held in the weights matrix to save space, i.e. weights is modified in place
    output_layer_add_1(weights, output_layer_temp_1);
    output_layer_add_2(output_layer_temp_1, output_layer_temp_2);
    output_layer_add_3(output_layer_temp_2, output_data);
    // output_layer_add(output_layer_temp_1, output_data);
}